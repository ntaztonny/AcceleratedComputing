#include "hip/hip_runtime.h"

//NotePad 1#################################################################################
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{
    /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  helloGPU<<<1,1>>>();
  hipDeviceSynchronize();
  helloCPU();
  helloGPU<<<1,1>>>();
  hipDeviceSynchronize();
  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
}

//NotePad 2 - Parrral thread#################################################################################
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */

  firstParallel<<<5,5>>>();
  hipDeviceSynchronize();

  /*
   * Some code is needed below so that the CPU will wait
   * for the GPU kernels to complete before proceeding.
   */

}

//NotePad 3 - ThreadIndex#################################################################################

#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  }
}

int main()
{
  /*
   * This is one possible execution context that will make
   * the kernel launch print its success message.
   */

  printSuccessForCorrectExecutionConfiguration<<<256, 1024>>>();

  /*
   * Don't forget kernel execution is asynchronous and you must
   * sync on its completion.
   */

  hipDeviceSynchronize();
}

//NotePad 4 - ForLoops for Acceleration #################################################################################

#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
  printf("This is iteration number %d\n", i);
 
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  loop<<<1, 10>>>();
  hipDeviceSynchronize();
}


//NotePad 5 - Accelerating a For Loop with Multiple Blocks of Threads¶#################################################################################

#include <stdio.h>

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                         
  {
    a[i] *= 2;
  }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 1000;
  int *a;

  size_t size = N * sizeof(int);

  /*
   * Refactor this memory allocation to provide a pointer
   * `a` that can be used on both the host and the device.
   */

  //a = (int *)malloc(size);
 
  hipMallocManaged(&a, size);
  init(a, N);
 
  size_t threads_per_block = 256;
  size_t number_of_blocks = (N + threads_per_block - 1);

  /*
   * This launch will not work until the pointer `a` is also
   * available to the device.
   */
   

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  /*
   * Refactor to free memory that has been allocated to be
   * accessed by both the host and the device.
   */

  hipFree(a);
}


//NotePad 6 - Mismatched data and threds¶#################################################################################

#include <stdio.h>

/*
 * Currently, `initializeElementsTo`, if executed in a thread whose
 * `i` is calculated to be greater than `N`, will try to access a value
 * outside the range of `a`.
 *
 * Refactor the kernel defintition to prevent our of range accesses.
 */

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < N)
      a[i] = initialValue;
}

int main()
{
  /*
   * Do not modify `N`.
   */

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  /*
   * Assume we have reason to want the number of threads
   * fixed at `256`: do not modify `threads_per_block`.
   */

  size_t threads_per_block = 256;

  /*
   * Assign a value to `number_of_blocks` that will
   * allow for a working execution configuration given
   * the fixed values for `N` and `threads_per_block`.
   */

  size_t number_of_blocks = (N + threads_per_block -1)/threads_per_block;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  /*
   * Check to make sure all values in `a`, were initialized.
   */

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      hipFree(a);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}

//Notepad 7 Dealing with Data Sets Larger then the Grid¶#################################################################################

#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int gridStride = gridDim.x * blockDim.x;
  
  for(int m = idx; m < N; m +=gridStride)
   {
    a[m] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  
  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 256;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}

//Notepad 8 Error handling in Cuda ¶#################################################################################

#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipError_t directCudaerror = hipMallocManaged(&a, size);
  
  if(directCudaerror != hipSuccess ) 
  printf("A direct call to cuda was made and the error detected is: %s\n", hipGetErrorString(directCudaerror));

  init(a, N);

  size_t threads_per_block = 2048;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  
  //get errors
  hipError_t err = hipGetLastError();
        
 //the CPU waits for GPU
  hipError_t asynchronizationError = hipDeviceSynchronize();
  
  if(err != hipSuccess)
      printf("Check out the error:%s\n", hipGetErrorString(err));
      
  if(asynchronizationError != hipSuccess)
      printf("Check out the cuda independent operation (async) errorerror:%s\n", hipGetErrorString(asynchronizationError));
      
  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}





















































